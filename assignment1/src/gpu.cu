
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <ctype.h>
#include <unistd.h>
#define FLOPS   1
#define IOPS    2

#define WRAP_MEM_ALLOC(W_N,W_TYPE)					\
  a = malloc(sizeof(W_TYPE)*W_N);					\
  hipMalloc(&d_a,W_N*sizeof(W_TYPE));					\
  a##W_TYPE = (W_TYPE *) a;						\
  for (i = 0; i < W_N; i++) {a##W_TYPE[i] = i; }

#define CUDA_BENCHMARK(CB_FUNC,CB_BLOCK,CB_THREAD,CB_TYPE,CB_NOPS)	\
  hipEventRecord(start);						\
  CB_FUNC<<<CB_BLOCK,CB_THREAD>>>((CB_TYPE)d_a,cores_count,CB_NOPS);	\
  hipEventRecord(stop);						\
  hipEventSynchronize(stop);						\
  hipEventElapsedTime(&elapsedTime, start, stop);

int 
gcd ( int a, int b )
{
  int c;
  while ( a != 0 ) {
    c = a; a = b%a;  b = c;
  }
  return b;
}


__global__ void VectorOpsInt(int * a, int n, int n_ops) {
  /* int i = threadIdx.x; */
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = 0;

  if (i < n) {
    for (j = 0; j < n_ops; j++) { // 2 operations
      // 3 ops by line: mul, add, and mv
      a[i] = a[i]*453+123;
      a[i] = a[i]*1234+5678;
      a[i] = a[i]*9876+54321;
      a[i] = a[i]*54321+12345;
      a[i] = a[i]*67890+9876;
      a[i] = a[i]*4321+7654;
      a[i] = a[i]*13579+24680;
      a[i] = a[i]*97531+86420;
      a[i] = a[i]*102938+4756;
    }
  }
}

__global__ void VectorOpsFloat(float * a, int n, int n_ops) {
  /* int i = threadIdx.x; */
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = 0;

  if (i < n) {
    for (j = 0; j < n_ops; j++) { // 2 operations
      // 3 ops by line: mul, add, and mv
      a[i] = a[i]*4.5321+1.2345;
      a[i] = a[i]*453.21+123.45;
      a[i] = a[i]*4532.1+1234.5;
      a[i] = a[i]*6.789+9.876;
      a[i] = a[i]*67.89+98.76;
      a[i] = a[i]*678.9+987.6;
      a[i] = a[i]*1.357+246.8;
      a[i] = a[i]*13.57+24.68;
      a[i] = a[i]*135.7+246.8;
    }
  }
}

int computeConfiguration(int* config) {
  /**
     Gets the device properties to know the maximal number of threads we can
     get running on the GPU and how to organize them in blocks.
     Returns an integer array A of size 2 where:
     - A[0] = number of blocks
     - A[1] = number of threads per block
  **/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  int n_proc = devProp.multiProcessorCount;
  int threads_per_proc = devProp.maxThreadsPerBlock;
  int threads_per_block = devProp.maxThreadsPerBlock;
  int max_threads = n_proc*threads_per_proc;

  /* Compute the greatest common denominator of the maximum number of threads we can run on the GPU and the number of threads we can run per block. Therefore, we can homogeneously distribute threads over blocks.*/
  int n_blocks = gcd(max_threads, threads_per_block);
  int n_threads = max_threads/n_blocks;

  config[0] = n_blocks;
  config[1] = n_threads;
  return max_threads;
}

int getCoresCount() {
  /**
     Gets the device number of cores.
  **/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  int n_proc = devProp.multiProcessorCount;
  return n_proc;
}

float gpu_speed_benchmark(int type, int n_operations) {
  void *a = NULL;
  void *d_a = NULL;
  float elapsedTime = 0;
  int config[2] = {0};

  // Initialize cuda timer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int cores_count = computeConfiguration(config);// getCoresCount();

  int i = 0;
  float * afloat = NULL;
  int * aint = NULL;

  // Allocate memory on host and on device.
  switch (type) {
  case FLOPS:
    WRAP_MEM_ALLOC(cores_count,float)
      hipMemcpy(d_a, a, cores_count*sizeof(float), hipMemcpyHostToDevice);
      break;
  case IOPS:
    WRAP_MEM_ALLOC(cores_count,int)
      hipMemcpy(d_a, a, cores_count*sizeof(int), hipMemcpyHostToDevice);
      break;
  default:
    break;
  }

  // Launch benchmark
  switch(type) {
  case IOPS:
    CUDA_BENCHMARK(VectorOpsInt,config[0],config[1],int*,n_operations)
      /* CUDA_BENCHMARK(VectorOpsInt,1,cores_count,int*,n_operations) */
      /* Copy the result to the host */
      hipMemcpy(a, d_a, cores_count*sizeof(int), hipMemcpyDeviceToHost);
    break;
  case FLOPS:
    CUDA_BENCHMARK(VectorOpsFloat,config[0],config[1],float*,n_operations)
      /* CUDA_BENCHMARK(VectorOpsFloat,1,cores_count,float*,n_operations) */
      /* Copy the result to the host */
      hipMemcpy(a, d_a, cores_count*sizeof(float), hipMemcpyDeviceToHost);
    break;
  default:
    break;
  }

  free(a);
  hipFree(d_a);
  return elapsedTime;
}

float gpu_bandwidth_benchmark(int block_size, int n_operations) {
  int i = 0;
  float elapsedTime = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  void * a = NULL;
  void * d_a = NULL;
  void * d_b = NULL;
  char * achar = NULL;

  WRAP_MEM_ALLOC(block_size,char)
  hipMalloc(&d_b, block_size*sizeof(char));
  hipMemcpy(d_a, a, block_size*sizeof(char), hipMemcpyHostToDevice); 

  hipEventRecord(start);
  for (i = 0; i < n_operations; i++) {
    hipMemcpy(d_b, d_a, block_size, hipMemcpyDeviceToDevice); 
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);  

  free(a);
  hipFree(d_a);
  return elapsedTime;;
}

void print_usage(char* argv[]) {
  fprintf(stderr, "Usage:\t%s -a speed [-f|-i] [-h] [-n N] [-o O]\n", argv[0]);
  fprintf(stderr, "\t%s -a bandwidth [-b B] [-n N] [-o O]\n", argv[0]);
  fprintf(stderr, "\nOptions: \n");
  fprintf(stderr, "\t-b B\tSize of the block to be allocated in memory [default:1024].\n");
  fprintf(stderr, "\t-f\tFlag for FLOPS benchmarking, used by default.\n");
  fprintf(stderr, "\t-h\tShow this help screen.\n");
  fprintf(stderr, "\t-i\tFlag for IOPS benchmarking.\n");
  fprintf(stderr, "\t-n N\tNumber of time the benchmark is repeated [default: 1].\n");
  fprintf(stderr, "\t-o O\tNumber of operations per loop [default: 1E5].\n");
}

int main(int argc, char* argv[]) {
  int n_repeats = 1;
  long n_operations = 10000;
  int type = FLOPS;
  long block_size = 1024;
  int c, i;
  float elapsedTime = 0;
  char * a_value = NULL;
  hipDeviceProp_t dev;
  hipGetDeviceProperties(&dev, 0);

  // Parsing the command line
  while ((c = getopt(argc, argv, "fhia:b:n:o:")) != -1)
    switch (c) {
    case 'a':
      a_value = optarg;
      if (strcmp(a_value, "speed") and strcmp(a_value, "bandwidth")) {
        print_usage(argv);
        exit(1);
      }
      break;
    case 'b':
      block_size = (long)(atof(optarg)); // use atof to allow exponents
      break;
    case 'f':
      type = FLOPS;
      break;
    case 'h':
      print_usage(argv);
      exit(0);
    case 'i':
      type = IOPS;
      break;
    case 'n': // number of type we repeat the benchmark
      n_repeats = atoi(optarg);
      break;
    case 'o': // number of operations per loop
      n_operations = (long)(atof(optarg)); // use atof to allow exponents
      break;
    case '?':
      print_usage(argv);
      return 1;
    default:
      abort ();
    }

  int config[2] = {0};
  int n_cores = computeConfiguration(config); //getCoresCount();
  if (strcmp(a_value, "speed") == 0) {
    for (i = 0; i < n_repeats; i++) {
      elapsedTime = gpu_speed_benchmark(type, n_operations)/1000;
      float speed = ((27+2)*n_operations*n_cores)/elapsedTime;
      printf("gpu,speed,%s,%s,%d,%d,%d,%d,%f,%f\n",
             dev.name, type == FLOPS ? "FLOPS":"IOPS",
	     n_cores,i, n_repeats, n_operations, elapsedTime, speed);
    }
  } else if (strcmp(a_value, "bandwidth") == 0) {
    for (i = 0; i < n_repeats; i++) {
      elapsedTime = gpu_bandwidth_benchmark(block_size, n_operations)/1000;
      float bandwidth = (block_size*n_operations)/elapsedTime;
      printf("gpu,bandwidth,%s,%d,%d,%d,%d,%f,%f\n",
             dev.name, i, n_repeats, block_size, n_operations, elapsedTime, bandwidth);
    }
  }
}
